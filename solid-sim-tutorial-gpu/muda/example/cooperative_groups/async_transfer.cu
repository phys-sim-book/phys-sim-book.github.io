#include "hip/hip_runtime.h"
#include <catch2/catch.hpp>
#include <muda/muda.h>
#include <muda/container.h>
#include <muda/cuda/cooperative_groups/memcpy_async.h>
#include <muda/cuda/hip/hip_cooperative_groups.h>

using namespace muda;
namespace cg = cooperative_groups;
void async_transfer(HostVector<int>& res, HostVector<int>& gt)
{
    DeviceVector<int> data(128, 1);
    Launch(2, 64)
        .apply(
            [data = data.viewer()] __device__() mutable
            {
                __shared__ int smem[64];
                auto           block = cg::this_thread_block();
                cg::memcpy_async(block,
                                 smem,
                                 &data(block.group_index().x * block.num_threads()),
                                 64 * sizeof(int));
                cg::wait(block);
                int gtid   = cg::this_grid().thread_rank();
                int btid   = block.thread_rank();
                smem[btid] = gtid;
                block.sync();
                cg::memcpy_async(block,
                                 &data(block.group_index().x * block.num_threads()),
                                 smem,
                                 64 * sizeof(int));
                cg::wait(block);
            })
        .wait();
    gt.resize(128, 1);
    for(size_t i = 0; i < 128; i++)
        gt[i] = i;
    res = data;
}

TEST_CASE("async_transfer", "[cooperative_groups]")
{
    HostVector<int> res, gt;
    async_transfer(res, gt);
    REQUIRE(res == gt);
};