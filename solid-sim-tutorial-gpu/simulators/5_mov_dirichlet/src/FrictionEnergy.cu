#include "hip/hip_runtime.h"
#include "FrictionEnergy.h"
#include <muda/muda.h>
#include <muda/container.h>
#include <stdio.h>
#include "device_uti.h"
#define epsv 1e-3

using namespace muda;

template <typename T, int dim>
struct FrictionEnergy<T, dim>::Impl
{
    DeviceBuffer<T> device_v;
    DeviceBuffer<T> device_mu_lambda;
    DeviceBuffer<T> device_grad;
    DeviceTripletMatrix<T, 1> device_hess;
    T hhat;
    Eigen::Matrix<T, dim, 1> n;
    int N;
};

template <typename T, int dim>
FrictionEnergy<T, dim>::FrictionEnergy() = default;

template <typename T, int dim>
FrictionEnergy<T, dim>::~FrictionEnergy() = default;

template <typename T, int dim>
FrictionEnergy<T, dim>::FrictionEnergy(FrictionEnergy<T, dim> &&rhs) = default;

template <typename T, int dim>
FrictionEnergy<T, dim> &FrictionEnergy<T, dim>::operator=(FrictionEnergy<T, dim> &&rhs) = default;

template <typename T, int dim>
FrictionEnergy<T, dim>::FrictionEnergy(const FrictionEnergy<T, dim> &rhs)
    : pimpl_{std::make_unique<Impl>(*rhs.pimpl_)} {}

template <typename T, int dim>
FrictionEnergy<T, dim>::FrictionEnergy(const std::vector<T> &v, T hhat, const std::vector<T> &n)
    : pimpl_{std::make_unique<Impl>()}
{
    pimpl_->N = v.size() / dim;
    pimpl_->device_v.copy_from(v);
    pimpl_->device_mu_lambda.resize(pimpl_->N);
    pimpl_->hhat = hhat;
    pimpl_->n = Eigen::Map<const Eigen::Matrix<T, dim, 1>>(n.data());
    pimpl_->device_grad.resize(pimpl_->N * dim);
    pimpl_->device_hess.resize_triplets(pimpl_->N * dim * dim);
    pimpl_->device_hess.reshape(v.size(), v.size());
}

template <typename T, int dim>
void FrictionEnergy<T, dim>::update_v(const DeviceBuffer<T> &v)
{
    pimpl_->device_v.view().copy_from(v);
}
template <typename T, int dim>
DeviceBuffer<T> &FrictionEnergy<T, dim>::get_mu_lambda()
{
    return pimpl_->device_mu_lambda;
}

template <typename T, int dim>
T __device__ FrictionEnergy<T, dim>::f0(T vbarnorm, T Epsv, T hhat)
{
    if (vbarnorm >= Epsv)
    {
        return vbarnorm * hhat;
    }
    else
    {
        T vbarnormhhat = vbarnorm * hhat;
        T epsvhhat = Epsv * hhat;
        return vbarnormhhat * vbarnormhhat * (-vbarnormhhat / 3.0 + epsvhhat) / (epsvhhat * epsvhhat) + epsvhhat / 3.0;
    }
}

template <typename T, int dim>
T __device__ FrictionEnergy<T, dim>::f1_div_vbarnorm(T vbarnorm, T Epsv)
{
    if (vbarnorm >= Epsv)
    {
        return 1.0 / vbarnorm;
    }
    else
    {
        return (-vbarnorm + 2.0 * Epsv) / (Epsv * Epsv);
    }
}

template <typename T, int dim>
T __device__ FrictionEnergy<T, dim>::f_hess_term(T vbarnorm, T Epsv)
{
    if (vbarnorm >= Epsv)
    {
        return -1.0 / (vbarnorm * vbarnorm);
    }
    else
    {
        return -1.0 / (Epsv * Epsv);
    }
}

template <typename T, int dim>
T FrictionEnergy<T, dim>::val()
{
    auto &device_v = pimpl_->device_v;
    auto &device_mu_lambda = pimpl_->device_mu_lambda;
    auto &hhat = pimpl_->hhat;
    auto &n = pimpl_->n;
    int N = device_v.size() / dim;
    DeviceBuffer<T> device_val(N);

    ParallelFor(256).apply(N, [device_val = device_val.viewer(), device_v = device_v.cviewer(), device_mu_lambda = device_mu_lambda.cviewer(), hhat, n, this] __device__(int i) mutable
                           {
        Eigen::Matrix<T, dim, dim> T_mat = Eigen::Matrix<T, dim, dim>::Identity() - n * n.transpose();
        if (device_mu_lambda(i) > 0)
        {
            Eigen::Matrix<T, dim, 1> v;
            for (int j = 0; j < dim; ++j)
            {
                v(j) = device_v(i * dim + j);
            }
            Eigen::Matrix<T, dim, 1> vbar = T_mat * v;
            T vbarnorm = vbar.norm();
            T val = f0(vbarnorm, epsv, hhat);
            device_val(i) = device_mu_lambda(i) * val;
        } })
        .wait();

    return devicesum(device_val);
}

template <typename T, int dim>
const DeviceBuffer<T> &FrictionEnergy<T, dim>::grad()
{
    auto &device_v = pimpl_->device_v;
    auto &device_mu_lambda = pimpl_->device_mu_lambda;
    auto &hhat = pimpl_->hhat;
    auto &n = pimpl_->n;
    int N = device_v.size() / dim;
    auto &device_grad = pimpl_->device_grad;
    device_grad.fill(0);

    ParallelFor(256).apply(N, [device_v = device_v.cviewer(), device_mu_lambda = device_mu_lambda.cviewer(), device_grad = device_grad.viewer(), hhat, n, this] __device__(int i) mutable
                           {
        Eigen::Matrix<T, dim, dim> T_mat = Eigen::Matrix<T, dim, dim>::Identity() - n * n.transpose();
        if (device_mu_lambda(i) > 0)
        {
            Eigen::Matrix<T, dim, 1> v;
            for (int j = 0; j < dim; ++j)
            {
                v(j) = device_v(i * dim + j);
            }
            Eigen::Matrix<T, dim, 1> vbar = T_mat * v;
            T vbarnorm = vbar.norm();
            T grad_factor = f1_div_vbarnorm(vbarnorm, epsv);
            Eigen::Matrix<T, dim, 1> grad = grad_factor * T_mat * vbar;

            for (int j = 0; j < dim; ++j)
            {
                device_grad(i * dim + j) = device_mu_lambda(i) * grad(j);
            }
        } })
        .wait();

    return device_grad;
}
template <typename T, int dim>
const DeviceTripletMatrix<T, 1> &FrictionEnergy<T, dim>::hess()
{
    auto &device_v = pimpl_->device_v;
    auto &device_mu_lambda = pimpl_->device_mu_lambda;
    auto &hhat = pimpl_->hhat;
    auto &n = pimpl_->n;
    auto &device_hess = pimpl_->device_hess;
    auto device_hess_row_idx = device_hess.row_indices();
    auto device_hess_col_idx = device_hess.col_indices();
    auto device_hess_val = device_hess.values();
    int N = device_v.size() / dim;
    device_hess_val.fill(0);
    ParallelFor(256).apply(N, [device_v = device_v.cviewer(), device_mu_lambda = device_mu_lambda.cviewer(), device_hess_val = device_hess_val.viewer(), device_hess_row_idx = device_hess_row_idx.viewer(), device_hess_col_idx = device_hess_col_idx.viewer(), hhat, n, N, this] __device__(int i) mutable
                           {
            for (int j = 0; j < dim; ++j)
            {
                for (int k = 0; k < dim; ++k)
                {
                    int idx = i * dim * dim + j * dim + k;
                    device_hess_row_idx(idx) = i * dim + j;
                    device_hess_col_idx(idx) = i * dim + k;
                }
            }
        Eigen::Matrix<T, dim, dim> T_mat = Eigen::Matrix<T, dim, dim>::Identity() - n * n.transpose();
        if (device_mu_lambda(i) > 0)
        {
            Eigen::Matrix<T, dim, 1> v;
            for (int j = 0; j < dim; ++j)
            {
                v(j) = device_v(i * dim + j);
            }
            Eigen::Matrix<T, dim, 1> vbar = T_mat * v;
            T vbarnorm = vbar.norm();
            Eigen::Matrix<T, dim, dim> inner_term = Eigen::Matrix<T, dim, dim>::Identity() * f1_div_vbarnorm(vbarnorm, epsv);
            if (vbarnorm != 0)
            {
                inner_term += f_hess_term(vbarnorm, epsv) / vbarnorm * vbar * vbar.transpose();
            }
            Eigen::Matrix<T, dim, dim> local_hess;
            make_PSD(inner_term, local_hess);
            local_hess = device_mu_lambda(i) * T_mat * local_hess * T_mat.transpose() / hhat;
            for (int j = 0; j < dim; ++j)
            {
                for (int k = 0; k < dim; ++k)
                {
                    int idx = i * dim * dim + j * dim + k;
                    device_hess_val(idx) = local_hess(j, k);
                }
            }
        } })
        .wait();
    return device_hess;
}
template class FrictionEnergy<float, 2>;
template class FrictionEnergy<float, 3>;
template class FrictionEnergy<double, 2>;
template class FrictionEnergy<double, 3>;
