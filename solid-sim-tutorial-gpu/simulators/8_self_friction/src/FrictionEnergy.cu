#include "hip/hip_runtime.h"
﻿#include "FrictionEnergy.h"
#include <muda/muda.h>
#include <muda/container.h>
#include <stdio.h>
#include "device_uti.h"
#define epsv 1e-3

using namespace muda;

template <typename T, int dim>
struct FrictionEnergy<T, dim>::Impl
{
    DeviceBuffer<T> device_v;
    DeviceBuffer<T> device_mu_lambda, device_mu_lambda_self, device_r_self;
    DeviceBuffer<T> device_grad;
    DeviceBuffer<int> device_bp, device_be;
    DeviceBuffer<Eigen::Matrix<T, 2, 1>> device_n_self;
    DeviceTripletMatrix<T, 1> device_hess;
    T hhat;
    Eigen::Matrix<T, dim, 1> n;
    int N, npe;
};

template <typename T, int dim>
FrictionEnergy<T, dim>::FrictionEnergy() = default;

template <typename T, int dim>
FrictionEnergy<T, dim>::~FrictionEnergy() = default;

template <typename T, int dim>
FrictionEnergy<T, dim>::FrictionEnergy(FrictionEnergy<T, dim> &&rhs) = default;

template <typename T, int dim>
FrictionEnergy<T, dim> &FrictionEnergy<T, dim>::operator=(FrictionEnergy<T, dim> &&rhs) = default;

template <typename T, int dim>
FrictionEnergy<T, dim>::FrictionEnergy(const FrictionEnergy<T, dim> &rhs)
    : pimpl_{std::make_unique<Impl>(*rhs.pimpl_)} {}

template <typename T, int dim>
FrictionEnergy<T, dim>::FrictionEnergy(const std::vector<T> &v, T hhat, const std::vector<T> &n, const std::vector<int> &bp, const std::vector<int> &be, int npe)
    : pimpl_{std::make_unique<Impl>()}
{
    pimpl_->N = v.size() / dim;
    pimpl_->npe = npe;
    pimpl_->device_v.copy_from(v);
    pimpl_->device_mu_lambda.resize(pimpl_->N);
    pimpl_->hhat = hhat;
    pimpl_->n = Eigen::Map<const Eigen::Matrix<T, dim, 1>>(n.data());
    pimpl_->device_grad.resize(pimpl_->N * dim);
    pimpl_->device_hess.resize_triplets(pimpl_->N * dim * dim + bp.size() * be.size() / 2 * 36);
    pimpl_->device_hess.reshape(v.size(), v.size());
    pimpl_->device_mu_lambda_self.resize(npe);
    pimpl_->device_bp.copy_from(bp);
    pimpl_->device_be.copy_from(be);
    pimpl_->device_n_self.resize(npe);
    pimpl_->device_r_self.resize(npe);
}

template <typename T, int dim>
void FrictionEnergy<T, dim>::update_v(const DeviceBuffer<T> &v)
{
    pimpl_->device_v.view().copy_from(v);
}
template <typename T, int dim>
DeviceBuffer<T> &FrictionEnergy<T, dim>::get_mu_lambda()
{
    return pimpl_->device_mu_lambda;
}
template <typename T, int dim>
DeviceBuffer<T> &FrictionEnergy<T, dim>::get_mu_lambda_self()
{
    return pimpl_->device_mu_lambda_self;
}
template <typename T, int dim>
DeviceBuffer<Eigen::Matrix<T, 2, 1>> &FrictionEnergy<T, dim>::get_n_self()
{
    return pimpl_->device_n_self;
}

template <typename T, int dim>
DeviceBuffer<T> &FrictionEnergy<T, dim>::get_r_self()
{
    return pimpl_->device_r_self;
}
template <typename T, int dim>
T __device__ FrictionEnergy<T, dim>::f0(T vbarnorm, T Epsv, T hhat)
{
    if (vbarnorm >= Epsv)
    {
        return vbarnorm * hhat;
    }
    else
    {
        T vbarnormhhat = vbarnorm * hhat;
        T epsvhhat = Epsv * hhat;
        return vbarnormhhat * vbarnormhhat * (-vbarnormhhat / 3.0 + epsvhhat) / (epsvhhat * epsvhhat) + epsvhhat / 3.0;
    }
}

template <typename T, int dim>
T __device__ FrictionEnergy<T, dim>::f1_div_vbarnorm(T vbarnorm, T Epsv)
{
    if (vbarnorm >= Epsv)
    {
        return 1.0 / vbarnorm;
    }
    else
    {
        return (-vbarnorm + 2.0 * Epsv) / (Epsv * Epsv);
    }
}

template <typename T, int dim>
T __device__ FrictionEnergy<T, dim>::f_hess_term(T vbarnorm, T Epsv)
{
    if (vbarnorm >= Epsv)
    {
        return -1.0 / (vbarnorm * vbarnorm);
    }
    else
    {
        return -1.0 / (Epsv * Epsv);
    }
}

template <typename T, int dim>
T FrictionEnergy<T, dim>::val()
{
    auto &device_v = pimpl_->device_v;
    auto &device_mu_lambda = pimpl_->device_mu_lambda;
    auto &hhat = pimpl_->hhat;
    auto &n = pimpl_->n;
    auto &device_bp = pimpl_->device_bp;
    auto &device_be = pimpl_->device_be;
    int Nbp = device_bp.size(), Nbe = device_be.size() / 2;
    int N = device_v.size() / dim;
    DeviceBuffer<T> device_val1(N);
    DeviceBuffer<T> device_val2(pimpl_->npe);
    ParallelFor(256).apply(N, [device_val1 = device_val1.viewer(), device_v = device_v.cviewer(), device_mu_lambda = device_mu_lambda.cviewer(), hhat, n, this] __device__(int i) mutable
                           {
        Eigen::Matrix<T, dim, dim> T_mat = Eigen::Matrix<T, dim, dim>::Identity() - n * n.transpose();
        if (device_mu_lambda(i) > 0)
        {
            Eigen::Matrix<T, dim, 1> v;
            for (int j = 0; j < dim; ++j)
            {
                v(j) = device_v(i * dim + j);
            }
            Eigen::Matrix<T, dim, 1> vbar = T_mat * v;
            T vbarnorm = vbar.norm();
            T val = f0(vbarnorm, epsv, hhat);
            device_val1(i) = device_mu_lambda(i) * val;
        } })
        .wait();
    ParallelFor(256).apply(pimpl_->npe, [device_val2 = device_val2.viewer(), device_v = device_v.cviewer(), device_mu_lambda_self = pimpl_->device_mu_lambda_self.cviewer(), device_n_self = pimpl_->device_n_self.cviewer(), device_r_self = pimpl_->device_r_self.cviewer(), device_bp = device_bp.cviewer(), device_be = device_be.cviewer(), Nbp, Nbe, hhat, this] __device__(int i) mutable
                           {
            if (device_mu_lambda_self(i) > 0)
            {
                int xI = device_bp(i / Nbe);
                int eI0 = device_be(2 * (i % Nbe)), eI1 = device_be(2 * (i % Nbe) + 1);
                Eigen::Vector<T, 2> vp, ve0, ve1;
                vp << device_v(xI * dim), device_v(xI * dim + 1);
                ve0 << device_v(eI0 * dim), device_v(eI0 * dim + 1);
                ve1 << device_v(eI1 * dim), device_v(eI1 * dim + 1);
    			Eigen::Matrix<T, 2, 2> T_mat = Eigen::Matrix<T, 2, 2>::Identity() - device_n_self(i) * device_n_self(i).transpose();
    			Eigen::Matrix<T, 2, 1> rel_v = vp - ((1 - device_r_self(i)) * ve0 + device_r_self(i) * ve1);
    			Eigen::Matrix<T, 2, 1> vbar = T_mat * rel_v;
    			device_val2(i) = device_mu_lambda_self(i) *f0(vbar.norm(), epsv, hhat);
            } })
        .wait();
    return devicesum(device_val1) + devicesum(device_val2);
}

template <typename T, int dim>
const DeviceBuffer<T> &FrictionEnergy<T, dim>::grad()
{
    auto &device_v = pimpl_->device_v;
    auto &device_mu_lambda = pimpl_->device_mu_lambda;
    auto &hhat = pimpl_->hhat;
    auto &n = pimpl_->n;
    int N = device_v.size() / dim;
    auto &device_grad = pimpl_->device_grad;
    auto &device_bp = pimpl_->device_bp;
    auto &device_be = pimpl_->device_be;
    int Nbp = device_bp.size(), Nbe = device_be.size() / 2;
    device_grad.fill(0);

    ParallelFor(256).apply(N, [device_v = device_v.cviewer(), device_mu_lambda = device_mu_lambda.cviewer(), device_grad = device_grad.viewer(), hhat, n, this] __device__(int i) mutable
                           {
        Eigen::Matrix<T, dim, dim> T_mat = Eigen::Matrix<T, dim, dim>::Identity() - n * n.transpose();
        if (device_mu_lambda(i) > 0)
        {
            Eigen::Matrix<T, dim, 1> v;
            for (int j = 0; j < dim; ++j)
            {
                v(j) = device_v(i * dim + j);
            }
            Eigen::Matrix<T, dim, 1> vbar = T_mat * v;
            T vbarnorm = vbar.norm();
            T grad_factor = f1_div_vbarnorm(vbarnorm, epsv);
            Eigen::Matrix<T, dim, 1> grad = grad_factor * T_mat * vbar;

            for (int j = 0; j < dim; ++j)
            {
                device_grad(i * dim + j) = device_mu_lambda(i) * grad(j);
            }
        } })
        .wait();
    ParallelFor(256).apply(pimpl_->npe, [device_v = device_v.cviewer(), device_mu_lambda_self = pimpl_->device_mu_lambda_self.cviewer(), device_n_self = pimpl_->device_n_self.cviewer(), device_r_self = pimpl_->device_r_self.cviewer(), device_bp = device_bp.cviewer(), device_be = device_be.cviewer(), device_grad = device_grad.viewer(), Nbp, Nbe, hhat, this] __device__(int i) mutable
                           {
        if (device_mu_lambda_self(i) > 0)
        {
            int xI = device_bp(i / Nbe);
            int eI0 = device_be(2 * (i % Nbe)), eI1 = device_be(2 * (i % Nbe) + 1);
            Eigen::Vector<T, 2> vp, ve0, ve1;
            vp << device_v(xI * dim), device_v(xI * dim + 1);
            ve0 << device_v(eI0 * dim), device_v(eI0 * dim + 1);
            ve1 << device_v(eI1 * dim), device_v(eI1 * dim + 1);
            Eigen::Matrix<T, 2, 2> Tmat = Eigen::Matrix<T, 2, 2>::Identity() - device_n_self(i) * device_n_self(i).transpose();
            Eigen::Matrix<T, 2, 1> rel_v = vp - ((1 - device_r_self(i)) * ve0 + device_r_self(i) * ve1);
            Eigen::Matrix<T, 2, 1> vbar = Tmat * rel_v;
            T grad_factor = f1_div_vbarnorm(vbar.norm(), epsv);
            Eigen::Matrix<T, 2, 1> grad = grad_factor * Tmat * vbar;
            for (int j = 0; j < dim; ++j)
            {
                atomic_add(&device_grad(xI * dim + j), device_mu_lambda_self(i) * grad(j));
                atomic_add(&device_grad(eI0 * dim + j), device_mu_lambda_self(i) * grad(j) * -(1 - device_r_self(i)));
                atomic_add(&device_grad(eI1 * dim + j), device_mu_lambda_self(i) * grad(j) * (-device_r_self(i)));
            }
        } })
        .wait();

    return device_grad;
}
template <typename T, int dim>
const DeviceTripletMatrix<T, 1> &FrictionEnergy<T, dim>::hess()
{
    auto &device_v = pimpl_->device_v;
    auto &device_mu_lambda = pimpl_->device_mu_lambda;
    auto &hhat = pimpl_->hhat;
    auto &n = pimpl_->n;
    auto &device_bp = pimpl_->device_bp;
    auto &device_be = pimpl_->device_be;
    int Nbp = device_bp.size(), Nbe = device_be.size() / 2;
    auto &device_hess = pimpl_->device_hess;
    auto device_hess_row_idx = device_hess.row_indices();
    auto device_hess_col_idx = device_hess.col_indices();
    auto device_hess_val = device_hess.values();
    int N = device_v.size() / dim;
    device_hess_val.fill(0);
    ParallelFor(256).apply(N, [device_v = device_v.cviewer(), device_mu_lambda = device_mu_lambda.cviewer(), device_hess_val = device_hess_val.viewer(), device_hess_row_idx = device_hess_row_idx.viewer(), device_hess_col_idx = device_hess_col_idx.viewer(), hhat, n, N, this] __device__(int i) mutable
                           {
        Eigen::Matrix<T, dim, dim> T_mat = Eigen::Matrix<T, dim, dim>::Identity() - n * n.transpose();
        for (int j = 0; j < dim; ++j)
        {
            for (int k = 0; k < dim; ++k)
            {
                int idx = i * dim * dim + j * dim + k;
                device_hess_row_idx(idx) = i * dim + j;
                device_hess_col_idx(idx) = i * dim + k;
            }
        }
        if (device_mu_lambda(i) > 0)
        {
            Eigen::Matrix<T, dim, 1> v;
            for (int j = 0; j < dim; ++j)
            {
                v(j) = device_v(i * dim + j);
            }
            Eigen::Matrix<T, dim, 1> vbar = T_mat * v;
            T vbarnorm = vbar.norm();
            Eigen::Matrix<T, dim, dim> inner_term = Eigen::Matrix<T, dim, dim>::Identity() * f1_div_vbarnorm(vbarnorm, epsv);
            if (vbarnorm != 0)
            {
                inner_term += f_hess_term(vbarnorm, epsv) / vbarnorm * vbar * vbar.transpose();
            }
            Eigen::Matrix<T, dim, dim> local_hess;
            make_PSD(inner_term, local_hess);
            local_hess = device_mu_lambda(i) * T_mat * local_hess * T_mat.transpose() / hhat;
            for (int j = 0; j < dim; ++j)
            {
                for (int k = 0; k < dim; ++k)
                {
                    int idx = i * dim * dim + j * dim + k;
                    device_hess_val(idx) = local_hess(j, k);
                }
            }
        } })
        .wait();
    ParallelFor(256).apply(pimpl_->npe, [N, device_v = device_v.cviewer(), device_mu_lambda_self = pimpl_->device_mu_lambda_self.cviewer(), device_n_self = pimpl_->device_n_self.cviewer(), device_r_self = pimpl_->device_r_self.cviewer(), device_hess_val = device_hess_val.viewer(), device_hess_row_idx = device_hess_row_idx.viewer(), device_hess_col_idx = device_hess_col_idx.viewer(), device_bp = device_bp.cviewer(), device_be = device_be.cviewer(), Nbp, Nbe, hhat, this] __device__(int i) mutable
                           {
                               int xI = device_bp(i / Nbe);
                               int eI0 = device_be(2 * (i % Nbe)), eI1 = device_be(2 * (i % Nbe) + 1);
                               int index[3] = {xI, eI0, eI1};
                               for (int nI = 0; nI < 3; ++nI)
                               {
                                   for (int nJ = 0; nJ < 3; ++nJ)
                                   {
                                       for (int c = 0; c < 2; ++c)
                                       {
                                           for (int r = 0; r < 2; ++r)
                                           {
                                               int idx = index[nI] * 2 + r;
                                               int jdx = index[nJ] * 2 + c;
                                               int kdx = N * dim * dim +i*36+ nI * 12 + nJ * 4 + c * 2 + r;
                                               device_hess_row_idx(kdx) = idx;
                                               device_hess_col_idx(kdx) = jdx;
                                           }
                                       }
                                   }
                               }
                               if (device_mu_lambda_self(i) > 0)
                               {

                                   Eigen::Vector<T, 2> vp, ve0, ve1;
                                   vp << device_v(xI * dim), device_v(xI * dim + 1);
                                   ve0 << device_v(eI0 * dim), device_v(eI0 * dim + 1);
                                   ve1 << device_v(eI1 * dim), device_v(eI1 * dim + 1);
                                   Eigen::Matrix<T, 2, 2> Tmat = Eigen::Matrix<T, 2, 2>::Identity() - device_n_self(i) * device_n_self(i).transpose();
                                   Eigen::Matrix<T, 2, 1> rel_v = vp - ((1 - device_r_self(i)) * ve0 + device_r_self(i) * ve1);
                                   Eigen::Matrix<T, 2, 1> vbar = Tmat* rel_v;
                                   T vbarnorm = vbar.norm();
                                   Eigen::Matrix<T, 2, 2> inner_term = Eigen::Matrix<T, 2, 2>::Identity() * f1_div_vbarnorm(vbarnorm, epsv);
                                   if (vbarnorm != 0)
                                   {
                                       inner_term += f_hess_term(vbarnorm, epsv) / vbarnorm * vbar * vbar.transpose();
                                   }
                                   Eigen::Matrix<T, 2, 2> local_hess;
                                   make_PSD(inner_term, local_hess);
                                   local_hess = device_mu_lambda_self(i) * Tmat * local_hess * Tmat.transpose() / hhat;

                                   T d_rel_v_dv[3] = {1, -(1 - device_r_self(i)), -device_r_self(i)};
                                   for (int nI = 0; nI < 3; ++nI)
                                   {
                                       for (int nJ = 0; nJ < 3; ++nJ)
                                       {
                                           for (int c = 0; c < 2; ++c)
                                           {
                                               for (int r = 0; r < 2; ++r)
                                               {
                                                   int kdx =N * dim * dim +i*36+ nI * 12 + nJ * 4 + c * 2 + r;
                                                   device_hess_val(kdx) = d_rel_v_dv[nI] * d_rel_v_dv[nJ] * local_hess(r, c);
                                               }
                                           }
                                       }
                                   }
                               } })
        .wait();
    return device_hess;
}
template class FrictionEnergy<float, 2>;
template class FrictionEnergy<float, 3>;
template class FrictionEnergy<double, 2>;
template class FrictionEnergy<double, 3>;
