#include "hip/hip_runtime.h"
#include <catch2/catch.hpp>
#include <muda/muda.h>
using namespace muda;

void log_test()
{
    Logger logger_;

    auto logger = logger_.viewer();
    Launch().apply([logger] __device__() mutable {  //
        logger << "threadIdx: " << threadIdx << "; blockIdx: " << blockIdx << "\n";
        int2 i2 = make_int2(1, 2);
        logger << "int2: " << i2 << "\n";
        float3 v3 = make_float3(1.0f, 2.0f, 3.0f);
        logger << "float3: " << v3 << "\n";

        LogProxy proxy{logger};
        int N[3] = {1, 2, 3};
        for(int i = 0; i < 3; ++i)
        {
            proxy << N[i] << " ";
        }
        proxy << "\n";
    });

    wait_device();
    logger_.retrieve(std::cout);
}

TEST_CASE("log_test", "[log]")
{
    log_test();
}
