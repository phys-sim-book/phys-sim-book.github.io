#include "device_uti.h"
using namespace muda;

template <typename T>
T devicesum(const DeviceBuffer<T> &buffer)
{
    T sum = 0.0f;                  // Result of the reduction
    T *d_out;                      // Device memory to store the result of the reduction
    hipMalloc(&d_out, sizeof(T)); // Allocate memory for the result

    // DeviceReduce is assumed to be part of the 'muda' library or similar
    DeviceReduce().Sum(buffer.data(), d_out, buffer.size());

    // Copy the result back to the host
    hipMemcpy(&sum, d_out, sizeof(T), hipMemcpyDeviceToHost);

    // Clean up
    hipFree(d_out);
    return sum;
}
template float devicesum<float>(const DeviceBuffer<float> &);
template double devicesum<double>(const DeviceBuffer<double> &);

template <typename T, int Size>
void __device__ make_PSD(const Eigen::Matrix<T, Size, Size> &hess, Eigen::Matrix<T, Size, Size> &PSD)
{
    Eigen::SelfAdjointEigenSolver<Eigen::Matrix<T, Size, Size>> eigensolver(hess);
    Eigen::Matrix<T, Size, 1> lam = eigensolver.eigenvalues();
    Eigen::Matrix<T, Size, Size> V = eigensolver.eigenvectors();
    // set all negative eigenvalues to zero
    Eigen::Matrix<T, Size, Size> lamDiag;
    lamDiag.setZero();
    for (int i = 0; i < Size; i++)
        if (lam(i) > 0)
            lamDiag(i, i) = lam(i);

    Eigen::Matrix<T, Size, Size> VT = V.transpose();

    PSD = V * lamDiag * VT;
}

template void __device__ make_PSD<float, 2>(const Eigen::Matrix<float, 2, 2> &hess, Eigen::Matrix<float, 2, 2> &PSD);
template void __device__ make_PSD<double, 2>(const Eigen::Matrix<double, 2, 2> &hess, Eigen::Matrix<double, 2, 2> &PSD);
template void __device__ make_PSD<float, 3>(const Eigen::Matrix<float, 3, 3> &hess, Eigen::Matrix<float, 3, 3> &PSD);
template void __device__ make_PSD<double, 3>(const Eigen::Matrix<double, 3, 3> &hess, Eigen::Matrix<double, 3, 3> &PSD);
template void __device__ make_PSD<float, 4>(const Eigen::Matrix<float, 4, 4> &hess, Eigen::Matrix<float, 4, 4> &PSD);
template void __device__ make_PSD<double, 4>(const Eigen::Matrix<double, 4, 4> &hess, Eigen::Matrix<double, 4, 4> &PSD);
template void __device__ make_PSD<float, 6>(const Eigen::Matrix<float, 6, 6> &hess, Eigen::Matrix<float, 6, 6> &PSD);
template void __device__ make_PSD<double, 6>(const Eigen::Matrix<double, 6, 6> &hess, Eigen::Matrix<double, 6, 6> &PSD);