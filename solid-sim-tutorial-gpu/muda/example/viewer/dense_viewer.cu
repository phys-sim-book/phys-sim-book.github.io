#include <catch2/catch.hpp>
#include <muda/muda.h>
#include <muda/container.h>
#include <muda/buffer.h>
#include <example_common.h>
using namespace muda;

void dense_viewer(HostVector<int>& ground_truth, HostVector<int>& res)
{
    example_desc(R"(an example for using dense viewer.)");

    hipSetDevice(0);

    DeviceVar<int> scalar = 2;
    // thrust device_vector
    DeviceVector<int> vector(32, 1);
    // muda device_buffer
    DeviceVector<int> result;
    result.resize(32, 1);

    ParallelFor(32 /*blockDim*/)
        .apply(32 /*count*/,
               [scalar = scalar.viewer(),  // the same as scalar = make_dense(scalar)
                vector = vector.viewer(),  // the same as vector = make_dense(vector)
                buffer = result.viewer()]  // the same as buffer = make_dense(buffer)
               __device__(int i) mutable { buffer(i) = scalar * vector(i); })
        .wait();
    res = result;
    ground_truth.resize(32, 2);
}

TEST_CASE("dense_viewer", "[viewer]")
{
    HostVector<int> ground_truth, res;
    dense_viewer(ground_truth, res);
    REQUIRE(ground_truth == res);
}
